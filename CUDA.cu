#include "hip/hip_runtime.h"
#include "CUDA.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hipblas.h"

#include <iostream>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>


// for debugging in GPU
#ifdef DEBUG_GPU
#define GpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line)
{

	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}
inline void memoryPrint(int line)
{
	float free_m, total_m, used_m;
	size_t free_t, total_t;
	hipMemGetInfo(&free_t, &total_t);
	free_m = (uint)free_t / 1048576.0;
	total_m = (uint)total_t / 1048576.0;
	used_m = total_m - free_m;
	printf("  line .... %d\tfree .... %f MB\ttotal ....%f MB\tused %f MB\n", line, free_m, total_m, used_m);
}
#else
#define GpuErrorCheck(ans) { ans; }
#endif

using namespace cv::cuda;

CUDA::CUDA(void)
{
}

CUDA::~CUDA(void)
{
}


/////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////
////////////////////// Device Code //////////////////////
/////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////

__global__ void make_PC_GPU(
	const cuda::PtrStepSz<uchar3> color_src,
	const cuda::PtrStepSz<ushort> depth_src,
	double scaleZ,
	double* K,
	double* R_wc_inv,
	double* t_wc,
	double* dev_x,
	double* dev_y,
	double* dev_z,
	uchar* dev_b,
	uchar* dev_g,
	uchar* dev_r)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int cols = color_src.cols;

	if (0 <= x && x < color_src.cols && 0 <= y && y < color_src.rows) {
		uchar3 color = color_src(y, x);
		dev_b[y * cols + x] = color.x;
		dev_g[y * cols + x] = color.y;
		dev_r[y * cols + x] = color.z;

		ushort depth_level = depth_src(y, x);

		double Z = depth_level_2_Z_s_direct(depth_level, scaleZ);

		double3 C_world = MVG(K, R_wc_inv, t_wc, x, y, Z);
		dev_x[y * cols + x] = C_world.x;
		dev_y[y * cols + x] = C_world.y;
		dev_z[y * cols + x] = C_world.z;
	}
}

__global__ void perform_projection_GPU(
	int ppc_size,
	int total_num_cameras,
	int cam_num,
	cuda::PtrStepSz<uchar3> proj_img,
	cuda::PtrStepSz<uchar> is_hole_proj_img,
	cuda::PtrStepSz<double> depth_value_img,
	double* dev_ProjMatrix,
	float* dev_x,
	float* dev_geo_y,
	float* dev_z,
	uchar* dev_color_y,
	uchar* dev_u,
	uchar* dev_v,
	bool* dev_occlusion)
{
	//////////////////////////////////
	/// 0		4		8		12 ///
	/// 1		5		9		13 ///
	/// 2		6		10		14 ///
	/// 3		7		11		15 ///
	//////////////////////////////////
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int _width = depth_value_img.cols, _height = depth_value_img.rows;
	if (0 <= i && i < ppc_size) {
		int offset = i * total_num_cameras;
		int proj_offset = 16 * cam_num;
		if (!dev_occlusion[offset + cam_num]) {
			// projetion_XYZ_2_UV
			double _u, _v, w;
			_u = dev_ProjMatrix[proj_offset + 0] * (double)dev_x[i] + dev_ProjMatrix[proj_offset + 4] * (double)dev_geo_y[i] + dev_ProjMatrix[proj_offset + 8] * (double)dev_z[i] + dev_ProjMatrix[proj_offset + 12];
			_v = dev_ProjMatrix[proj_offset + 1] * (double)dev_x[i] + dev_ProjMatrix[proj_offset + 5] * (double)dev_geo_y[i] + dev_ProjMatrix[proj_offset + 9] * (double)dev_z[i] + dev_ProjMatrix[proj_offset + 13];
			w = dev_ProjMatrix[proj_offset + 2] * (double)dev_x[i] + dev_ProjMatrix[proj_offset + 6] * (double)dev_geo_y[i] + dev_ProjMatrix[proj_offset + 10] * (double)dev_z[i] + dev_ProjMatrix[proj_offset + 14];

			_u /= w;
			_v /= w;

			int u = __double2int_rn(_u);
			int v = __double2int_rn(_v);

			double dist = find_point_dist(w, proj_offset, dev_ProjMatrix);

			if ((u < 0) || (v < 0) || (u > _width - 1) || (v > _height - 1)) return;

			if (depth_value_img(v, u) == -1) {
				depth_value_img(v, u) = dist;
				is_hole_proj_img(v, u) = 0;
			}
			else {
				if (dist < depth_value_img(v, u))
					depth_value_img(v, u) = dist;
				else
					return;
			}

			int location = offset + cam_num;
			proj_img(v, u) = make_uchar3(dev_color_y[location], dev_u[location], dev_v[location]);
			// if (u > _height - 1)
				// printf("cam: %d\tpoint: %d\tu: %d\tv: %d\n", cam_num, i, u, v);
		}
	}
}

__device__ double depth_level_2_Z_s_direct(ushort d, double scaleZ)
{
	return (double)d / scaleZ;
}

__device__ double3 MVG(
	double* K,
	double* R_wc_inv,
	double* t_wc,
	int x,
	int y,
	double Z)
{
	/////////////////////////
	/// 0		3		6 ///
	/// 1		4		7 ///
	/// 2		5		8 ///
	/////////////////////////
	double X_cam = (x - K[6]) * (Z / K[0]);
	double Y_cam = (y - K[7]) * (Z / K[4]);

	// cam coordinate
	double3 C_cam = make_double3(X_cam, Y_cam, Z);

	// assuming R, t as matrix world to cam
	C_cam.x -= t_wc[0];
	C_cam.y -= t_wc[1];
	C_cam.z -= t_wc[2];

	double3 C_world = make_double3(
		R_wc_inv[0] * C_cam.x + R_wc_inv[3] * C_cam.y + R_wc_inv[6] * C_cam.z,
		R_wc_inv[1] * C_cam.x + R_wc_inv[4] * C_cam.y + R_wc_inv[7] * C_cam.z,
		R_wc_inv[2] * C_cam.x + R_wc_inv[5] * C_cam.y + R_wc_inv[8] * C_cam.z);
	
	return C_world;
}

__device__ double find_point_dist(double w, int proj_offset, double* projMatrix)
{
	double numerator = 0., denominator = 0., dist = 0.;
	double M[3][3];
	for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
			M[i][j] = projMatrix[proj_offset + 4 * j + i];

	for (int i = 0; i < 3; i++)
		denominator = denominator + (M[2][i] * M[2][i]);

	denominator = sqrt(denominator);
	numerator = determinant(M);

	// sign
	if (numerator < 0) numerator = -1;
	else if (numerator == 0) numerator = 0;
	else numerator = 1;

	numerator = numerator * w;

	if (denominator == 0) {
		printf("Denominator Error\n");
	}
	else dist = (numerator / denominator);

	return dist;
}

__device__ double determinant(double mat[3][3])
{
	double D = 0;

	D = mat[0][0] * ((mat[1][1] * mat[2][2]) - (mat[2][1] * mat[1][2]))
		- mat[0][1] * (mat[1][0] * mat[2][2] - mat[2][0] * mat[1][2])
		+ mat[0][2] * (mat[1][0] * mat[2][1] - mat[2][0] * mat[1][1]);

	return D;
}

///////////////////////////////////////////////////////
///////////////////////////////////////////////////////
////////////////////// Host Code //////////////////////
///////////////////////////////////////////////////////
///////////////////////////////////////////////////////
void CUDA::make_PC(
	Mat color_img,
	Mat depth_img,
	int data_mode,
	double scaleZ,
	double* hst_K,
	double* hst_R_wc_inv,
	double* hst_t_wc,
	double* hst_x,
	double* hst_y,
	double* hst_z,
	uchar* hst_b,
	uchar* hst_g,
	uchar* hst_r)
{
	// TODO: develop to operate for mode 0~3.
	// For now, it can be operated correctly ONLY for mode 4 ~ 13.
	GpuMat color_img_gpu, depth_img_gpu;
	color_img_gpu.upload(color_img);
	depth_img_gpu.upload(depth_img);

	dim3 block(32, 8);
	dim3 grid(divUp(color_img_gpu.cols, block.x), divUp(color_img_gpu.rows, block.y));

	int numpix = color_img.rows * color_img.cols;
	double* dev_x, * dev_y, * dev_z;
	uchar* dev_b, * dev_g, * dev_r;
	double* dev_K, * dev_R_wc_inv, * dev_t_wc;

	GpuErrorCheck(hipMalloc(&dev_x, sizeof(double) * numpix));
	GpuErrorCheck(hipMalloc(&dev_y, sizeof(double) * numpix));
	GpuErrorCheck(hipMalloc(&dev_z, sizeof(double) * numpix));
	GpuErrorCheck(hipMalloc(&dev_b, sizeof(uchar) * numpix));
	GpuErrorCheck(hipMalloc(&dev_g, sizeof(uchar) * numpix));
	GpuErrorCheck(hipMalloc(&dev_r, sizeof(uchar) * numpix));
	GpuErrorCheck(hipMalloc(&dev_K, sizeof(double) * 9));
	GpuErrorCheck(hipMalloc(&dev_R_wc_inv, sizeof(double) * 9));
	GpuErrorCheck(hipMalloc(&dev_t_wc, sizeof(double) * 3));

	GpuErrorCheck(hipMemcpy(dev_K, hst_K, sizeof(double) * 9, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_R_wc_inv, hst_R_wc_inv, sizeof(double) * 9, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_t_wc, hst_t_wc, sizeof(double) * 3, hipMemcpyHostToDevice));

	make_PC_GPU << < grid, block >> > (color_img_gpu, depth_img_gpu, scaleZ, dev_K, dev_R_wc_inv, dev_t_wc, dev_x, dev_y, dev_z, dev_b, dev_g, dev_r);
	hipDeviceSynchronize();

	GpuErrorCheck(hipMemcpy(hst_x, dev_x, sizeof(double) * numpix, hipMemcpyDeviceToHost));
	GpuErrorCheck(hipMemcpy(hst_y, dev_y, sizeof(double) * numpix, hipMemcpyDeviceToHost));
	GpuErrorCheck(hipMemcpy(hst_z, dev_z, sizeof(double) * numpix, hipMemcpyDeviceToHost));
	GpuErrorCheck(hipMemcpy(hst_b, dev_b, sizeof(uchar) * numpix, hipMemcpyDeviceToHost));
	GpuErrorCheck(hipMemcpy(hst_g, dev_g, sizeof(uchar) * numpix, hipMemcpyDeviceToHost));
	GpuErrorCheck(hipMemcpy(hst_r, dev_r, sizeof(uchar) * numpix, hipMemcpyDeviceToHost));

	GpuErrorCheck(hipFree(dev_x));
	GpuErrorCheck(hipFree(dev_y));
	GpuErrorCheck(hipFree(dev_z));
	GpuErrorCheck(hipFree(dev_b));
	GpuErrorCheck(hipFree(dev_g));
	GpuErrorCheck(hipFree(dev_r));
	GpuErrorCheck(hipFree(dev_K));
	GpuErrorCheck(hipFree(dev_R_wc_inv));
	GpuErrorCheck(hipFree(dev_t_wc));

	color_img_gpu.release();
	depth_img_gpu.release();
}

void CUDA::perform_projection(
	Mat sample_mat,
	uchar* proj_data,
	uchar* is_hole_proj_data,
	double* depth_value_data,
	int total_num_cameras,
	double* hst_ProjMatrix,
	int ppc_size,
	float* hst_x,
	float* hst_geo_y,
	float* hst_z,
	uchar* hst_color_y,
	uchar* hst_u,
	uchar* hst_v,
	bool* hst_occlusion)
{
	int threadsPerBlock = 1024;
	int blocksPerGrid =	divUp(ppc_size + threadsPerBlock - 1, threadsPerBlock);
	float* dev_x, * dev_geo_y, * dev_z;
	uchar* dev_color_y, * dev_u, * dev_v;
	bool* dev_occlusion;
	double* dev_ProjMatrix;
	size_t total_size = ppc_size * total_num_cameras;
	
	printf("total size: %u\n", total_size);
	clock_t start = clock();
	GpuErrorCheck(hipMalloc(&dev_x, sizeof(float) * ppc_size));
	GpuErrorCheck(hipMalloc(&dev_geo_y, sizeof(float) * ppc_size));
	GpuErrorCheck(hipMalloc(&dev_z, sizeof(float) * ppc_size));
	GpuErrorCheck(hipMalloc(&dev_color_y, sizeof(uchar) * total_size));
	GpuErrorCheck(hipMalloc(&dev_u, sizeof(uchar) * total_size));
	GpuErrorCheck(hipMalloc(&dev_v, sizeof(uchar) * total_size));
	GpuErrorCheck(hipMalloc(&dev_occlusion, sizeof(bool) * total_size));
	GpuErrorCheck(hipMalloc(&dev_ProjMatrix, sizeof(double) * 16 * total_num_cameras));

	GpuErrorCheck(hipMemcpy(dev_x, hst_x, sizeof(float) * ppc_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_geo_y, hst_geo_y, sizeof(float) * ppc_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_z, hst_z, sizeof(float) * ppc_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_color_y, hst_color_y, sizeof(uchar) * total_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_u, hst_u, sizeof(uchar) * total_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_v, hst_v, sizeof(uchar) * total_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_occlusion, hst_occlusion, sizeof(bool) * total_size, hipMemcpyHostToDevice));
	GpuErrorCheck(hipMemcpy(dev_ProjMatrix, hst_ProjMatrix, sizeof(double) * 16 * total_num_cameras, hipMemcpyHostToDevice));
	clock_t end = clock();

	int rows = sample_mat.rows;
	int cols = sample_mat.cols;
	start = clock();
	GpuMat proj_img_gpu, hole_img_gpu, depth_img_gpu;

	for (int cam_num = 0; cam_num < total_num_cameras; ++cam_num) {
		Mat proj_img(rows, cols, CV_8UC3, Scalar(0, 0, 0));
		Mat hole_img(rows, cols, CV_8UC1, Scalar(1));
		Mat depth_img(rows, cols, CV_64FC1, Scalar(-1.0));

		proj_img_gpu.upload(proj_img);
		hole_img_gpu.upload(hole_img);
		depth_img_gpu.upload(depth_img);

		perform_projection_GPU <<< blocksPerGrid, threadsPerBlock >>> (ppc_size, total_num_cameras, cam_num, proj_img_gpu, hole_img_gpu, depth_img_gpu, dev_ProjMatrix, dev_x, dev_geo_y, dev_z, dev_color_y, dev_u, dev_v, dev_occlusion);
		hipDeviceSynchronize();

		proj_img_gpu.download(proj_img);
		hole_img_gpu.download(hole_img);
		depth_img_gpu.download(depth_img);

		size_t global_offset = cam_num * rows * cols;
		for (int y = 0; y < rows; ++y) {
			uchar* proj_ptr = proj_img.ptr<uchar>(y);
			uchar* hole_ptr = hole_img.ptr<uchar>(y);
			double* depth_ptr = depth_img.ptr<double>(y);
			for (int x = 0; x < cols; ++x) {
				size_t offset = global_offset + (y * cols + x);
				proj_data[offset * 3 + 0] = proj_ptr[x * 3 + 0];
				proj_data[offset * 3 + 1] = proj_ptr[x * 3 + 1];
				proj_data[offset * 3 + 2] = proj_ptr[x * 3 + 2];
				is_hole_proj_data[offset] = hole_ptr[x];
				depth_value_data[offset] = depth_ptr[x];
			}
		}
	}
		
	end = clock();
	printf("GPU computation time: %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	
	GpuErrorCheck(hipFree(dev_x));
	GpuErrorCheck(hipFree(dev_geo_y));
	GpuErrorCheck(hipFree(dev_z));
	GpuErrorCheck(hipFree(dev_color_y));
	GpuErrorCheck(hipFree(dev_u));
	GpuErrorCheck(hipFree(dev_v));
	GpuErrorCheck(hipFree(dev_occlusion));
	GpuErrorCheck(hipFree(dev_ProjMatrix));

	proj_img_gpu.release();
	hole_img_gpu.release();
	depth_img_gpu.release();
}

void CUDA::test()
{
	Mat img = imread("logo.png");
	GpuMat img_gpu;

	img_gpu.upload(img);
	img_gpu.download(img);

	imwrite("save.png", img);
}